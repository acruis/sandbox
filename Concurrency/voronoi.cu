#include "hip/hip_runtime.h"
// ---------------
// CS3211 CUDA Lab
// ---------------

__device__ __host__ int distance(int x1, int y1, int x2, int y2) 
{
	return (x1 - x2) * (x1 - x2) + (y1 - y2) * (y1 - y2); 
}

/* The CPU version of Voronoi Diagram computation */ 
void cpuVoronoiDiagram(Point2D *points, int *output, int noPoints, int width, int height) 
{
	int id = 0; 

	for (int j = 0; j < height; j++) 
		for (int i = 0; i < width; i++) {
			int best = 0; 
			int minDist = distance(i, j, points[0].x, points[0].y);

			for (int k = 1; k < noPoints; k++) {
				int myDist = distance(i, j, points[k].x, points[k].y); 

				if (myDist < minDist) {
					minDist = myDist; 
					best = k;
				}
			}

			output[id++] = best; 
		}
}

/* GPU Version of Voronoi diagram computation 
 * Naive implementation
 */
__global__ void kernelNaiveVoronoiDiagram(Point2D *points, int *result, int noPoints, int width) 
{
	int threadX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int threadY = (blockIdx.y * blockDim.y) + threadIdx.y;

	int best = 0;
	int minDist = distance(threadX, threadY, points[0].x, points[0].y);

	for (int i = 1; i < noPoints; i++) {
		int myDist = distance(threadX, threadY, points[i].x, points[i].y);

		if (myDist < minDist) {
			minDist = myDist;
			best = i;
		}
	}

	result[(threadY * width) + threadX] = best;
}

/* GPU Version of Voronoi diagram computation
 * Using shared memory
 * We assume for simplicity that noPoints is divisible by TILE
 */
__global__ void kernelSharedVoronoiDiagram(Point2D *points, int *result, int noPoints, int width) 
{
	int threadX = (blockIdx.x * blockDim.x) + threadIdx.x;
	int threadY = (blockIdx.y * blockDim.y) + threadIdx.y;

	int best = 0;
	int minDist = distance(threadX, threadY, points[0].x, points[0].y);

	__shared__ Point2D interPoints[TILE];

	for (int m = 0; m < noPoints; m += TILE) {
		int interPointIdx = (threadIdx.y * blockDim.x) + threadIdx.x;
		int pointIdx = m + interPointIdx;
		interPoints[interPointIdx] = points[pointIdx];

		__syncthreads();

		for (int k = 0; k < TILE; k++) {
			int myDist = distance(threadX, threadY, interPoints[k].x, interPoints[k].y);

			if (myDist < minDist) {
				minDist = myDist;
				best = m + k;
			}
		}

		__syncthreads();
	}

	result[(threadY * width) + threadX] = best;
}

